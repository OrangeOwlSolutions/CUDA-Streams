#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Utilities.cuh"

using namespace std;

#define NUM_THREADS 32
#define NUM_BLOCKS 16
#define NUM_STREAMS 3

__global__ void kernel(const int *in, int *out, int N)
{
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int end =  N;
    for (int i = start; i < end; i += blockDim.x * gridDim.x)
    {
        out[i] = in[i] * in[i];
    }
}

int main()
{
    const int N = 6000000;
    
    // --- Host side input data allocation and initialization. Registering host memory as page-locked (required for asynch hipMemcpyAsync).
    int *h_in = new int[N];	for(int i = 0; i < N; i++) h_in[i] = 5;
    gpuErrchk(hipHostRegister(h_in, N * sizeof(int), hipHostRegisterPortable));

    // --- Host side input data allocation and initialization. Registering host memory as page-locked (required for asynch hipMemcpyAsync).
    int *h_out = new int[N]; for(int i = 0; i < N; i++) h_out[i] = 0;
	gpuErrchk(hipHostRegister(h_out, N * sizeof(int), hipHostRegisterPortable));

    // --- Host side check results vector allocation and initialization
    int *h_checkResults = new int[N]; for(int i = 0; i < N; i++) h_checkResults[i] = h_in[i] * h_in[i];

    // --- Device side input data allocation.
    int *d_in = 0;				gpuErrchk(hipMalloc((void **)&d_in, N * sizeof(int)));

    // --- Device side output data allocation. 
	int *d_out = 0;				gpuErrchk( hipMalloc((void **)&d_out, N * sizeof(int)));
    
	int streamSize = N / NUM_STREAMS;
    size_t streamMemSize = N * sizeof(int) / NUM_STREAMS;

	// --- Set kernel launch configuration
    dim3 nThreads		= dim3(NUM_THREADS,1,1);
    dim3 nBlocks		= dim3(NUM_BLOCKS, 1,1);
    dim3 subKernelBlock = dim3((int)ceil((float)nBlocks.x / 2));

    // --- Create CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamCreate(&streams[i]));

	/**************************/
	/* BREADTH-FIRST APPROACH */
	/**************************/
	
	int offset = 0;
	hipMemcpyAsync(&d_in[offset], &h_in[offset], streamMemSize, hipMemcpyHostToDevice,     0);
	for(int i = 1; i < NUM_STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_in[offset], &h_in[offset], streamMemSize, hipMemcpyHostToDevice,     streams[i]);
	}

    kernel<<<subKernelBlock, nThreads>>>(&d_in[offset], &d_out[offset],   streamSize/2);
    kernel<<<subKernelBlock, nThreads>>>(&d_in[offset + streamSize/2],    &d_out[offset +  streamSize/2], streamSize/2);

	for(int i = 1; i < NUM_STREAMS; i++)
    {
        int offset = i * streamSize;
        kernel<<<subKernelBlock, nThreads, 0, streams[i]>>>(&d_in[offset], &d_out[offset],   streamSize/2);
        kernel<<<subKernelBlock, nThreads, 0, streams[i]>>>(&d_in[offset + streamSize/2],    &d_out[offset +  streamSize/2], streamSize/2);
    }

	for(int i = 1; i < NUM_STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&h_out[offset], &d_out[offset], streamMemSize, hipMemcpyDeviceToHost,   streams[i]);
	}

	hipMemcpyAsync(&h_out[offset], &d_out[offset], streamMemSize, hipMemcpyDeviceToHost,   0);
	for(int i = 1; i < NUM_STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&h_out[offset], &d_out[offset], streamMemSize, hipMemcpyDeviceToHost,   0);
	}

	for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamSynchronize(streams[i]));

    gpuErrchk(hipDeviceSynchronize());

    // --- Release resources
    gpuErrchk(hipHostUnregister(h_in));
    gpuErrchk(hipHostUnregister(h_out));
    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));

    for(int i = 0; i < NUM_STREAMS; i++)
        gpuErrchk(hipStreamDestroy(streams[i]));

    hipDeviceReset();  

    // --- GPU output check
    int sum = 0;
    for(int i = 0; i < N; i++)      
        sum += h_checkResults[i] - h_out[i];

    cout << "Error between CPU and GPU: " << sum << endl;

    delete[] h_in;
    delete[] h_out;
    delete[] h_checkResults;

    return 0;
}
